#include "hip/hip_runtime.h"
#include "spiker_network_gpu.h"
#include "spiker_network.h"
#include "utilities/text_formatter.h"
#include ""
#include <stdlib.h> //exit

// nvcc does not seem to like variadic macros, so we have to define
// one for each kernel parameter list:
// TL;DR: This is for IDE to supress formatting errors
#ifdef __HIPCC__
#define KERNEL_ARGS2(grid, block) <<< grid, block >>>
#define KERNEL_ARGS3(grid, block, sh_mem) <<< grid, block, sh_mem >>>
#define KERNEL_ARGS4(grid, block, sh_mem, stream) <<< grid, block, sh_mem, stream >>>
#else
#define KERNEL_ARGS2(grid, block)
#define KERNEL_ARGS3(grid, block, sh_mem)
#define KERNEL_ARGS4(grid, block, sh_mem, stream)
#endif

unsigned int GPU_READY = 0;
Neuron *gpu_neurons;
char* live_spike_array_gpu;
__device__ float SPIKE_VOLTAGE_GPU;
__device__ int MAX_NEURON_INPUTS_GPU;
__device__ float step_time_gpu = 1.0f;

__global__ void update_neuron(Neuron *neurons, unsigned int max_index, unsigned int min_index)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= max_index || index < min_index)
    {
        return;
    }
    float I = 0;
    int inputIdx, latencyBit;
    for (int i = 0; i < neurons[index].incomming_connections; i++)
    {
        inputIdx = neurons[index].inputs[i] - 1; // 0-based index afterwards
        latencyBit = 1 << neurons[index].latencies[i];
        // If there is spike with specific latency
        if (neurons[inputIdx].spike_train & latencyBit) {
            I += neurons[index].weights[i] * 1.6f; // TODO: FIXME remove value
        }
    }

    //v = v + time_delta * (0.04 * v^2 + 5 * v + 140 - u + I)
    // TODO: Validate mathematics (code review)!!!
    /*float core_calculation = 0.04f * neurons[index].v * neurons[index].v + 5.0f * neurons[index].v + 140.0f - neurons[index].u + I;
    core_calculation *= step_time_gpu;
    neurons[index].v += core_calculation;*/
    // TODO: fix hardcoded time
    neurons[index].v = neurons[index].v + step_time_gpu * (0.04f * neurons[index].v * neurons[index].v + 5.0f * neurons[index].v + 140.0f - neurons[index].u + I);
    // u = u + time_delta * (a * (b * v - u));
    // TODO: Validate mathematics (code review)!!!
    neurons[index].u = neurons[index].u + (step_time_gpu * (neurons[index].a * (neurons[index].b * neurons[index].v - neurons[index].u)));
    //printf("Spike voltrage %f \n", SPIKE_VOLTAGE_GPU);
    if (neurons[index].v >= SPIKE_VOLTAGE_GPU)
    {
        neurons[index].v = neurons[index].c;
        neurons[index].u = neurons[index].u + neurons[index].d;
        neurons[index].spike_train |= 1 << 0; // Set the first bit to 1 because the neuron spiked
    }
    return;
}

__global__ void move_to_next_step(Neuron* neurons, unsigned int max_index, unsigned int min_index)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= max_index)
    {
        return;
    }
    neurons[index].spike_train <<= 1;
}

__global__ void gather_spike_info(Neuron* neurons, char* spike_array, unsigned int max_index)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index >= max_index)
    {
        return;
    }
    if (neurons[index].spike_train & 1 << 1)
    {
        // If anyone wondering... yes, I wrote this function myself
        spike_array[index] = 0xFF;
    }
    else
    {
        spike_array[index] = 0x0;
    }
    //spike_array[index] = 0xFF;
    return;
}

extern "C" int check_cuda_failure(hipError_t err, const char * message)
{
    if (err != hipSuccess)
    {
        print_error("CUDA failure ");
        printf("(%s) Message: %s\n",message, hipGetErrorString(err));
        exit(1);
    }
}

extern "C" int init_gpu_network()
{
    hipMemcpyToSymbol((const void*)&MAX_NEURON_INPUTS_GPU, &max_connections, sizeof(max_connections));
    //hipMemcpy(&MAX_NEURON_INPUTS_GPU, &max_connections, sizeof(max_connections), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "MAX_INPUTS");
    hipMemcpyToSymbol((const void*)&SPIKE_VOLTAGE_GPU, &SPIKE_VOLTAGE, sizeof(SPIKE_VOLTAGE));
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "SPIKE_VOLGATE");
    hipMalloc(&gpu_neurons, sizeof(Neuron) * (main_neuron_count + input_neurons));
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "Malloc GPU neurons");
    hipMalloc(&live_spike_array_gpu, sizeof(char) * (main_neuron_count + input_neurons));
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "Malloc GPU spike array");
    hipMemcpy(gpu_neurons, neurons, sizeof(Neuron) * (main_neuron_count + input_neurons), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "Copy GPU neurons");

    
    GPU_READY = 1;
    print_success("GPU network initialized\n");
    return 0;
}

extern "C" int simulate_gpu_step()
{
    if (!GPU_READY)
    {
        return 1;
    }
    // TODO: Implement atomic block or mutex of some sort for other processes to probe neurons and insert data
    move_to_next_step KERNEL_ARGS2(main_neuron_count / GPU_BLOCKS + 1, GPU_BLOCKS)(gpu_neurons, main_neuron_count, input_neurons);
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "Shifting spikes");
    update_neuron KERNEL_ARGS2(main_neuron_count / GPU_BLOCKS + 1, GPU_BLOCKS)(gpu_neurons, main_neuron_count, input_neurons);
    hipDeviceSynchronize();
    check_cuda_failure(hipGetLastError(), "Neuron update");
    // TODO: Optimize (do not copy each time)
    //hipMemcpy(neurons, gpu_neurons, sizeof(Neuron) * main_neuron_count, hipMemcpyDeviceToHost);
    return 0;
}

int refresh_gpu_inputs_from_cpu()
{
    // TODO: Implement locking mechanism
    hipMemcpy(gpu_neurons, neurons, sizeof(Neuron) * input_neurons, hipMemcpyHostToDevice);
    return 0;
}

int transfer_gpu_spike_array_to_cpu()
{
    if (!GPU_READY)
    {
        return 1;
    }
    gather_spike_info KERNEL_ARGS2(main_neuron_count / GPU_BLOCKS + 1, GPU_BLOCKS)(gpu_neurons, live_spike_array_gpu, main_neuron_count);
    hipDeviceSynchronize(); 
    hipMemcpy(live_spike_array_cpu, live_spike_array_gpu, sizeof(char) * (input_neurons + main_neuron_count), hipMemcpyDeviceToHost);
    return 0;
}

extern "C" void free_gpu_network()
{
    hipFree(gpu_neurons);
    hipFree(live_spike_array_gpu);
    GPU_READY = 0;
    print_success("GPU network freed\n");
    return;
}
